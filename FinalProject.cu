#include "hip/hip_runtime.h"
#include <stdio.h>
#include <chrono>
#include <ctime>
#include <math.h>
#include <random>
#include <iostream>
#include <vector>


__global__ 
void solvePerceptron(double* weight1, double* weight2, double* weight3, double* weight4, double* b, double* throughput1, double* throughput2, double* throughput3, double* throughput4, double* out) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    double weightFunc = weight1[id]*throughput1[id] + weight2[id]*throughput2[id] + weight3[id]*throughput3[id] + weight4[id]*throughput4[id] + b[id];
    out[id] = 1/(1+exp(-weightFunc));
}

double calcDeltak(double yj, double yk, double dk, double wkj) {
    return yk*(1-yk)*(dk-yk);
}

double calcDeltaj(double yj, double yk, double dk, double wkj, double deltak) {
    return yj*(1-yj)*wkj*deltak;
}

double updateWeight(double w, double x_or_y, double eta, double delta) {
    return w+eta*delta*x_or_y;
}

int runIterations(double eta, double* possible_inputs, double* input_ans, double* error, double* weights1, double* weights2, double* weights3, double* weights4, double* layer2_weights, double* bias, double* layer2Bias) {
    double error_max = 1;
    std::vector<int> iteration_vector;
    double layer2Per = 0;
    //TODO: Send weights to CUDA GPU
    for (int i=1; i<16; ++i) iteration_vector.push_back(i);
    while (error_max > .05) {
        //Randomize order of input
        std::random_shuffle ( iteration_vector.begin(), iteration_vector.end() );
        for(int i = 0; i < iteration_vecotr.size(); i++) {
            //TODO: Solve Perceptrons Layer using CUDA Simutaneously
            //TODO: Calc Deltas
            //TODO: Update Weights
            //Calc Error for current input
            error[iteration_vector[i]] = input_ans[iteration_vector[i]]-layer2Per;
        }
        //TODO: CALC Max Error After iterations
    }
    return 0;
}

double* generateWeights(int numWeights) {
    double* weights = (double*)calloc(numWeights, sizeof(double));
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 1.0);
    for(int i = 0; i < numWeights; i++) {
        weights[i] = dis(gen);
    }
    return weights;
}

int main(int argc, char** argv)
{
    double* weights1 = generateWeights(4);
    double* weights2 = generateWeights(4);
    double* weights3 = generateWeights(4);
    double* weights4 = generateWeights(4);
    double* layer2_weights = generateWeights(4);
                                               
    double* bias = generateWeights(4);
                                               
    double* layer2Bias = generateWeights(1);
                                               
    double error [16] = {1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};
                                               
    double eta = .5;
                                               
    double possible_inputs[16][4] = {{0,0,0,0}, {0,0,0,1}, {0,0,1,0}, {0,0,1,1}, {0,1,0,0}, {0,1,0,1}, {0,1,1,0}, {0,1,1,1}, {1,0,0,0}, {1,0,0,1}, {1,0,1,0}, {1,0,1,1}, {1,1,0,0}, {1,1,0,1}, {1,1,1,0}, {1,1,1,1}};
                                               
    double input_ans[16] = {0,1,1,0,1,0,0,1,1,0,0,1,0,1,1,0};
    
    return 0;
}